#include "hip/hip_runtime.h"
#include "basicParallel.h"
using namespace std;

//o(1)
__global__ void normKernel(Point* d_basePointCloud, Point* d_targetPointCloud, float* d_normArray){
    int indexBase = blockIdx.x*blockDim.x+threadIdx.x;
    int indexTarget = blockIdx.y*blockDim.y+threadIdx.y;
    d_normArray[indexBase*CLOUDSIZE+indexTarget] = norm3d(d_basePointCloud[indexBase].x-d_targetPointCloud[indexTarget].x,\
                                                          d_basePointCloud[indexBase].y-d_targetPointCloud[indexTarget].y,\
                                                          d_basePointCloud[indexBase].z-d_targetPointCloud[indexTarget].z);
}

//o(n)
__global__ void minKernel(float* d_normArray, float* d_minArray){
    int indexBase = blockIdx.x*blockDim.x+threadIdx.x;
    float min = 1;
    for (int i = 0; i < CLOUDSIZE; i++)
        if(d_normArray[indexBase*CLOUDSIZE+i]<min) min = d_normArray[indexBase*CLOUDSIZE+i];
    d_minArray[indexBase] = min;
}

//o(logn)
__global__ void sumKernel(float* d_minArray, float* d_basicParaResList){
    __shared__ float partialSum[2*PARTIAlBLOCKSIZE];
    unsigned int t = threadIdx.x;
    unsigned int start = 2*blockIdx.x*blockDim.x;
    //loading data from global memory to share memory
    partialSum[t] = d_minArray[start+t];
    partialSum[blockDim.x+t] = d_minArray[start + blockDim.x+t];
    //compute
    for(unsigned int stride = 1; stride <= blockDim.x; stride <<= 1){
        __syncthreads();
        //if t % stride == 0 the thread will run the computation
        if(t % stride == 0)
            partialSum[2*t] += partialSum[2*t+stride];
    } 
    if(t==0) d_basicParaResList[blockIdx.x] = partialSum[0];
}


void basicParaCompute(Point* basePointcloud, Point* targetPointcloud, float* basicParaRes, int dev){
    hipSetDevice(dev);
    Point* d_basePointcloud, *d_targetPointcloud;
    float* d_normArray, *d_minArray, *d_basicParaRes, *d_basicParaResList;
    float* normArray, *minArray, *resList;
    int size = CLOUDSIZE*sizeof(Point);

    normArray = (float *)malloc(CLOUDSIZE*CLOUDSIZE*sizeof(float));
    minArray = (float *)malloc(CLOUDSIZE*sizeof(float));
    resList = (float *)malloc((CLOUDSIZE/PARTIAlBLOCKSIZE+1)*sizeof(float));

    //allocate device memory
    //and mv basePointcloud and target Pointcloud to device memory
    hipMalloc((void **)&d_basePointcloud, size);
    hipMemcpy(d_basePointcloud, basePointcloud, size, hipMemcpyHostToDevice);
    hipMalloc((void **)&d_targetPointcloud, size);
    hipMemcpy(d_targetPointcloud, targetPointcloud, size, hipMemcpyHostToDevice);
    //normArray and result space
    hipMalloc((void **)&d_normArray, CLOUDSIZE*CLOUDSIZE*sizeof(float));
    hipMalloc((void **)&d_minArray, CLOUDSIZE*sizeof(float));
    hipMalloc((void **)&d_basicParaResList, (CLOUDSIZE/PARTIAlBLOCKSIZE+1)*sizeof(float));
    hipMalloc((void **)&d_basicParaRes, sizeof(float));

    //kernel
    //compute 2-Norm
    dim3 normDimGrid(CLOUDSIZE/16+1,CLOUDSIZE/16+1,1);
    dim3 normDimBlock(16,16,1);
    normKernel <<< normDimGrid, normDimBlock >>> (d_basePointcloud,d_targetPointcloud,d_normArray);
    hipMemcpy(normArray, d_normArray,CLOUDSIZE*CLOUDSIZE*sizeof(float), hipMemcpyDeviceToHost);

    //compute min of norm array
    dim3 minDimGrid(CLOUDSIZE/16+1,1,1);
    dim3 minDimBlock(16,1,1);
    minKernel <<< minDimGrid, minDimBlock >>> (d_normArray, d_minArray);
    hipMemcpy(minArray, d_minArray, CLOUDSIZE*sizeof(float), hipMemcpyDeviceToHost);

    //compute sum
    dim3 sumDimGrid(CLOUDSIZE/PARTIAlBLOCKSIZE+1,1,1);
    dim3 sumDimBlock(PARTIAlBLOCKSIZE,1,1);
    sumKernel <<< sumDimGrid, sumDimBlock>>> (d_minArray, d_basicParaResList);
    hipMemcpy(resList, d_basicParaResList, (CLOUDSIZE/PARTIAlBLOCKSIZE+1)*sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < CLOUDSIZE/PARTIAlBLOCKSIZE+1; i++)
    {
        *basicParaRes+=resList[i];
    }
    
    //get result from device
    *basicParaRes /= CLOUDSIZE;
    
    //free memory
    hipFree(d_basePointcloud);
    hipFree(d_targetPointcloud);
    hipFree(d_normArray);
    hipFree(d_minArray);
    hipFree(d_basicParaRes);
    free(normArray);
    free(minArray);
}